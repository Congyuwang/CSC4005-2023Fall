#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h> // CUDA Header
#include "utils.hpp"

const int FILTER_SIZE = 3;
__constant__ float filter[FILTER_SIZE][FILTER_SIZE] = {
  { 1.0 / 9, 1.0 / 9, 1.0 / 9 },
  { 1.0 / 9, 1.0 / 9, 1.0 / 9 },
  { 1.0 / 9, 1.0 / 9, 1.0 / 9 }
};

/**
 * Sequentailly compute a single px.
 */
__global__ void
smooth_single_px(const unsigned char* input_buf,
                 unsigned char* output_buf,
                 const int row_length,
                 const int width,
                 const int height,
                 const int channel)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // keep the border
int h = idx / width;
  int w = idx % width;
  if (w >= width - 2 || h >= height - 2) {
    return;
  }

  idx *= channel;
  int mid_idx = idx + row_length;
  int out_idx = mid_idx + channel;
  int bot_idx = mid_idx + row_length;

  unsigned char cv_l_t_r = input_buf[idx + 0];
  unsigned char cv_l_t_g = input_buf[idx + 1];
  unsigned char cv_l_t_b = input_buf[idx + 2];
  unsigned char cv_m_t_r = input_buf[idx + 3];
  unsigned char cv_m_t_g = input_buf[idx + 4];
  unsigned char cv_m_t_b = input_buf[idx + 5];
  unsigned char cv_r_t_r = input_buf[idx + 6];
  unsigned char cv_r_t_g = input_buf[idx + 7];
  unsigned char cv_r_t_b = input_buf[idx + 8];

  unsigned char cv_l_m_r = input_buf[mid_idx + 0];
  unsigned char cv_l_m_g = input_buf[mid_idx + 1];
  unsigned char cv_l_m_b = input_buf[mid_idx + 2];
  unsigned char cv_m_m_r = input_buf[mid_idx + 3];
  unsigned char cv_m_m_g = input_buf[mid_idx + 4];
  unsigned char cv_m_m_b = input_buf[mid_idx + 5];
  unsigned char cv_r_m_r = input_buf[mid_idx + 6];
  unsigned char cv_r_m_g = input_buf[mid_idx + 7];
  unsigned char cv_r_m_b = input_buf[mid_idx + 8];

  unsigned char cv_l_b_r = input_buf[bot_idx + 0];
  unsigned char cv_l_b_g = input_buf[bot_idx + 1];
  unsigned char cv_l_b_b = input_buf[bot_idx + 2];
  unsigned char cv_m_b_r = input_buf[bot_idx + 3];
  unsigned char cv_m_b_g = input_buf[bot_idx + 4];
  unsigned char cv_m_b_b = input_buf[bot_idx + 5];
  unsigned char cv_r_b_r = input_buf[bot_idx + 6];
  unsigned char cv_r_b_g = input_buf[bot_idx + 7];
  unsigned char cv_r_b_b = input_buf[bot_idx + 8];

  int sum_r = cv_l_t_r * filter[0][0] + cv_m_t_r * filter[0][1] +
              cv_r_t_r * filter[0][2] + cv_l_m_r * filter[1][0] +
              cv_m_m_r * filter[1][1] + cv_r_m_r * filter[1][2] +
              cv_l_b_r * filter[2][0] + cv_m_b_r * filter[2][1] +
              cv_r_b_r * filter[2][2];

  int sum_g = cv_l_t_g * filter[0][0] + cv_m_t_g * filter[0][1] +
              cv_r_t_g * filter[0][2] + cv_l_m_g * filter[1][0] +
              cv_m_m_g * filter[1][1] + cv_r_m_g * filter[1][2] +
              cv_l_b_g * filter[2][0] + cv_m_b_g * filter[2][1] +
              cv_r_b_g * filter[2][2];

  int sum_b = cv_l_t_b * filter[0][0] + cv_m_t_b * filter[0][1] +
              cv_r_t_b * filter[0][2] + cv_l_m_b * filter[1][0] +
              cv_m_m_b * filter[1][1] + cv_r_m_b * filter[1][2] +
              cv_l_b_b * filter[2][0] + cv_m_b_b * filter[2][1] +
              cv_r_b_b * filter[2][2];

  output_buf[out_idx + 0] = static_cast<unsigned char>(sum_r);
  output_buf[out_idx + 1] = static_cast<unsigned char>(sum_g);
  output_buf[out_idx + 2] = static_cast<unsigned char>(sum_b);
}

int main(int argc, char** argv)
{
    // Verify input argument format
    if (argc != 3)
    {
        std::cerr << "Invalid argument, should be: ./executable "
                     "/path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }
    // Read from input JPEG
    const char* input_filepath = argv[1];
    std::cout << "Input file from: " << input_filepath << "\n";
    auto input_jpeg = read_from_jpeg(input_filepath);
    // Allocate memory on host (CPU)
    auto filteredImage = new unsigned char[input_jpeg.width * input_jpeg.height *
                                           input_jpeg.num_channels];
    // Allocate memory on device (GPU)
    unsigned char* d_input;
    unsigned char* d_output;
    hipMalloc((void**)&d_input, input_jpeg.width * input_jpeg.height *
                                 input_jpeg.num_channels * sizeof(unsigned char));
    hipMalloc((void**)&d_output, input_jpeg.width * input_jpeg.height *
                                  input_jpeg.num_channels * sizeof(unsigned char));
    // Initilize output image
    hipMemset((void**)&d_output, 0, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels * sizeof(unsigned char));
    // Copy input data from host to device
    hipMemcpy(d_input, input_jpeg.buffer, input_jpeg.width * input_jpeg.height *
                                           input_jpeg.num_channels * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    // Computation: Smoothing
    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int blockSize = 512;
    int numBlocks = (input_jpeg.width * (input_jpeg.height - 2)) / blockSize + 1;
    int row_length = input_jpeg.width * input_jpeg.num_channels;
    hipEventRecord(start, 0); // GPU start time
    smooth_single_px<<<numBlocks, blockSize>>>(
        d_input,
        d_output,
        row_length,
        input_jpeg.width,
        input_jpeg.height,
        input_jpeg.num_channels
    );
    hipEventRecord(stop, 0); // GPU end time
    hipEventSynchronize(stop);
    // Print the result of the GPU computation
    hipEventElapsedTime(&gpuDuration, start, stop);
    // Copy output data from device to host
    hipMemcpy(filteredImage, d_output, input_jpeg.width * input_jpeg.height *
                                        input_jpeg.num_channels * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    // Save output JPEG image
    const char* output_filepath = argv[2];
    std::cout << "Output file to: " << output_filepath << "\n";
    JPEGMeta output_jpeg{ filteredImage,
                          input_jpeg.width,
                          input_jpeg.height,
                          input_jpeg.num_channels,
                          input_jpeg.color_space };
    if (write_to_jpeg(output_jpeg, output_filepath)) {
      std::cerr << "Failed to write output JPEG\n";
      return -1;
    }
    // Release allocated memory on device and host
    hipFree(d_input);
    hipFree(d_output);
    delete[] input_jpeg.buffer;
    delete[] filteredImage;
    std::cout << "Transformation Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
