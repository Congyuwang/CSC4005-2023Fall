#include "hip/hip_runtime.h"
#include "../matrix_cuda.hpp"
#include <hip/hip_runtime.h> // CUDA Header
#include <iostream>

#define TILE 8

#define CUDA_ERR(val) check((val), #val, __FILE__, __LINE__)

void
check(hipError_t err, const char* const func, const char* const file, int const line)
{
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    throw std::exception();
  }
}

// CUDA kernel functon.
//
// !!! ASSUME ALL DIMS PADDED !!!
//
// Each block compute a tile of the result matrix.
//
// Requires 2D `TILE * TILE` THREADS_PER_BLOCK.
//
__global__ void
mat_mul(int* mat1, int* mat2, int* matr, size_t m, size_t k, size_t n)
{

  // CUDA L1 Cache
  __shared__ int mat1_tile[TILE][TILE];
  __shared__ int mat2_tile[TILE][TILE];

  int sum = 0;

  // 2D block and 2D thread
  // Each thread computes one cell in mat_3.
  size_t blkx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t blky = blockIdx.y * blockDim.y + threadIdx.y;

  for (size_t kk = 0; kk < k; kk += TILE) {

    // load M1 (blkx.., kk..)
    size_t j = kk + threadIdx.y;
    mat1_tile[threadIdx.x][threadIdx.y] = mat1[blkx * k + j];

    // load M2 (kk.., blky..)
    size_t i = kk + threadIdx.x;
    mat2_tile[threadIdx.x][threadIdx.y] = mat2[i * n + blky];

    __syncthreads();

    // compute result M1 (blkx.., kk..) @ M2 (kk.., blky..)
    for (size_t k = 0; k < TILE; ++k) {
      sum += mat1_tile[threadIdx.x][k] * mat2_tile[k][threadIdx.y];
    }

    __syncthreads();
  }

  matr[blkx * n + blky] = sum;
}

/// @brief matrix multiplication with CUDA core.
/// @param matrix1 !!PADDED
/// @param matrix2 !!PADDED
/// @return output matrix
Matrix
matrix_multiply(const Matrix& matrix1, const Matrix& matrix2)
{
  if (matrix1.getCols() != matrix2.getRows()) {
    std::cerr << "Matrix dimensions are not compatible for multiplication."
              << std::endl;
    throw std::exception();
  }

  const size_t M = matrix1.getRows(), K = matrix1.getCols(),
               N = matrix2.getCols();
  const size_t MT = matrix1.getRowsPadded(), KT = matrix1.getColsPadded(),
               NT = matrix2.getColsPadded();
  const size_t M1_SIZE = MT * KT, M2_SIZE = KT * NT, MR_SIZE = MT * NT;

  const dim3 THREADS_PER_BLOCK(TILE, TILE);
  const dim3 BLOCKS_PER_GRID(MT / TILE, NT / TILE);

  Matrix result(M, N, TILE, TILE);

  int* mat1;
  int* mat2;
  int* matr;
  CUDA_ERR(hipMalloc((void**)&mat1, M1_SIZE * sizeof(int)));
  CUDA_ERR(hipMalloc((void**)&mat2, M2_SIZE * sizeof(int)));
  CUDA_ERR(hipMalloc((void**)&matr, MR_SIZE * sizeof(int)));
  // Initilize result matrix
  CUDA_ERR(hipMemset(matr, 0, MR_SIZE * sizeof(int)));
  // Copy input matrix
  CUDA_ERR(hipMemcpy(
    mat1, matrix1.raw(), M1_SIZE * sizeof(int), hipMemcpyHostToDevice));
  CUDA_ERR(hipMemcpy(
    mat2, matrix2.raw(), M2_SIZE * sizeof(int), hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  float gpuDuration;
  CUDA_ERR(hipEventCreate(&start));
  CUDA_ERR(hipEventCreate(&stop));

  CUDA_ERR(hipEventRecord(start, 0)); // GPU start time
  mat_mul<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(mat1, mat2, matr, MT, KT, NT);
  CUDA_ERR(hipEventRecord(stop, 0)); // GPU end time

  // Print time of the GPU computation
  CUDA_ERR(hipEventSynchronize(stop));
  CUDA_ERR(hipEventElapsedTime(&gpuDuration, start, stop));
  // Copy output data from device to host
  CUDA_ERR(hipMemcpy(
    result.raw(), matr, MR_SIZE * sizeof(int), hipMemcpyDeviceToHost));

  std::cout << "Multiplication Complete!" << std::endl;
  std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds"
            << std::endl;

  return result;
}

int
main(int argc, char** argv)
{
  // Verify input argument format
  if (argc != 4) {
    std::cerr << "Invalid argument, should be: ./executable "
                 "/path/to/matrix1 /path/to/matrix2 /path/to/multiply_result"
              << std::endl;
    throw std::exception();
  }

  const std::string matrix1_path = argv[1];

  const std::string matrix2_path = argv[2];

  const std::string result_path = argv[3];

  Matrix matrix1 = Matrix::loadFromFile(matrix1_path, TILE, TILE);

  Matrix matrix2 = Matrix::loadFromFile(matrix2_path, TILE, TILE);

  Matrix result = matrix_multiply(matrix1, matrix2);

  result.saveToFile(result_path);

  std::cout << "Output file to: " << result_path << std::endl;

  return 0;
}
